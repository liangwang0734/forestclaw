#include "hip/hip_runtime.h"
__device__ void rpn2adv_cuda(int iface, int meqn,int mwaves, int maux,
     double ql[], double qr[], double auxl[], double auxr[],
     double wave[],double s[], double amdq[], double apdq[])
{
    int iface;

    wave[0][0] = qr[0] - ql[0];
    s[0] = auxr[iface];
    amdq[0] = min(auxr[iface], 0) * wave[1][1];
    apdq[0] = max(auxr[iface], 0) * wave[1][1];
}