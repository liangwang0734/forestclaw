#include "hip/hip_runtime.h"
__global__  void cudaclaw5_step2_GPU(int mx,int my,int meqn,int maux,int mbc,
                                     double *qold, double *aux, double dx,
                                     double dy,double dt, double cflgrid,
                                     double fm[], double fp[], 
                                     double gm[],double gp[], 
                                     cudaclaw5_rpn2_t cuda_rpn2,
                                     cudaclaw5_rpt2_t cuda_rpt2, 
                                     cudaclaw5_flux2_t cuda_flux2)
{

    int i,j
    double precision dtdx,dtdy,cfl1d

    cflgrid = 0.0;
    dtdx = dt/dx;
    dtdy = dt/dy;

#if 0   
    /* Allocate on GPU; Set to zero */
    fm = 0.d0
    fp = 0.d0
    gm = 0.d0
    gp = 0.d0
#endif    


}
