#include "hip/hip_runtime.h"
__global__ void cudaclaw5_step2_wrap(int maxm, int meqn, int maux, int mbc,
                                     int method[], int mthlim[], int mcapa, 
                                     int mwaves, int mx, int my, 
                                     double qold[],
                                     double aux[], 
                                     double dx, double dy, double dt,
                                     double cfl, double xlower, double ylower, 
                                     int level, double t, 
                                     cudaclaw5_rpn2_t rpn2,
                                     cudaclaw5_rpt2_t rpt2, 
                                     cudaclaw5_flux2_t flux2,
                                     int block_corner_count[], int ierror)
{
    double dtdx, dtdy
    int i,j,m

    /* Needed by Riemann solvers.  This should be fixed later by 
       a 'context' for a Riemann solver. */

    ierror = 0;


    /* GPU call : Distribute grid to blocks */
    int m = (mx+3);
    int n = (my+3); 
    dim3 block(32,32);  
    dim3 grid((m+block.x-1)/block.x,(n+block.y-1)/block.y);

    cudaclaw5_step2_GPU<<grid,block>>(mx,my,meqn,maux,mbc,
                                      qold_dev,aux_dev,
                                      dx,dy,dt,cfl,
                                      fm_dev,fp_dev,gm_dev,gp_dev,
                                      cuda_rpn2,rpt2,ierror);

    /* update q */
    dtdx = dt/dx
    dtdy = dt/dy
    for(m = 0; m < meqn; m++)
    {
        for(i = 0; i < mx; i++)
        {            
            for(j = 0; j < my; j++)
            {
               if (mcapa == 0)
               {
                  /* no capa array.  Standard flux differencing: */
                  qold[m][i][j] = qold[m][i][j]
                        - dtdx * (fm[m][i+1][j] - fp[m][i][j])
                        - dtdy * (gm[m][i][j+1] - gp[m][i][j]);
                }
                else if 
                {
                    /* with capa array. */
                    qold[m][i][j] = qold[m][i][j]
                                -(dtdx*(fm[m][i+1][j] - fp[m][i][j])
                                + dtdy*(gm[m][i][j+1] - gp[m][i][j]))/aux[mcapa][i][j];
               }
           }
        }
    }
}