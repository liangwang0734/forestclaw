#include "hip/hip_runtime.h"
/*
  Copyright (c) 2018 Carsten Burstedde, Donna Calhoun, Melody Shih, Scott Aiton, 
  Xinsheng Qin.
  All rights reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are met:

  * Redistributions of source code must retain the above copyright notice, this
  list of conditions and the following disclaimer.
  * Redistributions in binary form must reproduce the above copyright notice,
  this list of conditions and the following disclaimer in the documentation
  and/or other materials provided with the distribution.

  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
  DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
  FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
  DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
  SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
  OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/



#include "../fc2d_cudaclaw_cuda.h"

#include <cub/block/block_reduce.cuh>  

#include "cudaclaw_allocate.h"  /* Needed to for definition of 'fluxes' */

/* Include this here so we don't include device code in fc2d_cudaclaw_cuda.h */
__device__ double cudaclaw_limiter(int lim_choice, double r);

static
__device__
void cudaclaw_flux2_and_update(int mx, int my, int meqn, int mbc,
                               int maux, int mwaves, int mwork,
                               double xlower, double ylower, 
                               double dx, double dy,
                               double* qold, double* aux, 
                               double* fm, double* fp, 
                               double* gm, double* gp,
                               double* amdq_trans, double* apdq_trans, 
                               double* bmdq_trans, double* bpdq_trans,
                               double* waves, double *speeds,
                               double * maxcflblocks,
                               cudaclaw_cuda_rpn2_t rpn2,
                               cudaclaw_cuda_rpt2_t rpt2,
                               cudaclaw_cuda_b4step2_t b4step2,
                               int* order, int* mthlim,
                               double t,double dt)
{
    /* Does this 128 have to match the 128 grid size used to launch this kernel? */
    typedef hipcub::BlockReduce<double,FC2D_CUDACLAW_BLOCK_SIZE> BlockReduce;

    __shared__ typename BlockReduce::TempStorage temp_storage;

    extern __shared__ double shared_mem[];
    double* start = shared_mem + mwork*threadIdx.x;
    double* ql   = start;             //meqn
    double* qr   = ql+meqn;           //meqn
    double* qd   = qr+meqn;           //meqn
    double* auxl = qd+meqn;           //maux
    double* auxr = auxl+maux;         //maux
    double* auxd = auxr+maux;         //maux
    double* s    = auxd+maux;         //mwaves
    double* wave = s+mwaves;          //meqn*mwaves
    double* amdq = wave+meqn*mwaves;  //meqn
    double* apdq = amdq+meqn;         //meqn
    double* bmdq = apdq+meqn;         //meqn
    double* bpdq = bmdq+meqn;         //meqn
    double* aux1 = bpdq+meqn;         //2*maux
    double* aux2 = aux1+2*maux;       //2*maux
    double* aux3 = aux2+2*maux;       //2*maux
    double* bmasdq = aux3+2*maux;     //meqn
    double* bpasdq = bmasdq+meqn;     //meqn

    int mq, mw, m, k;
    int xs, ys, zs;
    int I, I_q, I_aux, I_waves, I_speeds;
    int thread_index;
    int ix,iy,ifaces_x, ifaces_y, num_ifaces;

    int i,j; /* Used for (i,j) indexing in patches  */
    double dtdx, dtdy;
    double maxcfl, cfl;
    double wnorm2,dotr,dotl, wlimitr,r;
    double cqxx;
    double cqyy;
    double gupdate;
    int imp;

    /* --------------------------------- Start code ----------------------------------- */

    ifaces_x = mx+2*mbc-1;
    ifaces_y = my+2*mbc-1;
    num_ifaces = ifaces_x*ifaces_y;

    dtdx = dt/dx;
    dtdy = dt/dy;

    /* Compute strides */
    xs = 1;
    ys = (2*mbc + mx)*xs;
    zs = (2*mbc + my)*xs*ys;

    maxcfl = 0;

    if (b4step2 != NULL)
    {
        for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
        {
            ix = thread_index % ifaces_x;
            iy = thread_index/ifaces_y;

            I = (iy + mbc-1)*ys + (ix + mbc-1)*xs;

            if (ix < mx + 2*mbc-1 && iy < my + 2*mbc-1)
            {
                for(mq = 0; mq < meqn; mq++)
                {
                    I_q = I + mq*zs;
                    qr[mq] = qold[I_q];  
                }

                for(m = 0; m < maux; m++)
                {
                    /* In case aux is already set */
                    I_aux = I + m*zs;
                    auxr[m] = aux[I_aux];
                }                        
                
                i = ix-(mbc-2);  /* i,j for index in the grid */
                j = iy-(mbc-2);
                b4step2(mbc,mx,my,meqn,qr,xlower,ylower,dx,dy, 
                        t,dt,maux,auxr,i,j);

                for(m = 0; m < maux; m++)
                {
                    /* In case aux is set by b4step2 */
                    I_aux = I + m*zs;
                    aux[I_aux] = auxr[m];
                }
            } /* Thread conditional */
        } /* Thread loop */

        __syncthreads();

    } /* b4step2 conditional */


    /* --------------------------- Normal solver : X faces ---------------------------- */
    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    {
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (iy + mbc-1)*ys + (ix + mbc-1)*xs;

        if (ix < mx + 2*mbc-1 && iy < my + 2*mbc-1)
        {
            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                ql[mq] = qold[I_q - 1];    /* Left  */
                qr[mq] = qold[I_q];        /* Right */
            }

            for(m = 0; m < maux; m++)
            {
                /* How is this getting set? */
                I_aux = I + m*zs;
                auxl[m] = aux[I_aux - 1];
                auxr[m] = aux[I_aux];
            }                        

            rpn2(0, meqn, mwaves, maux, ql, qr, auxl, auxr, wave, s, amdq, apdq);

            for (mq = 0; mq < meqn; mq++) 
            {
                I_q = I + mq*zs;
                fp[I_q] = -apdq[mq]; 
                fm[I_q] = amdq[mq];
                if (order[0] > 0)
                {
                    apdq_trans[I_q] = apdq[mq];  
                    amdq_trans[I_q] = amdq[mq];                    
                }
            }

            for (m = 0; m < meqn*mwaves; m++)
            {
                I_waves = I + m*zs;
                waves[I_waves] = wave[m];
            }                

            for (mw = 0; mw < mwaves; mw++)
            {
                I_speeds = I + mw*zs;
                speeds[I_speeds] = s[mw];
                cfl = abs(s[mw]*dtdx);
                if (cfl > maxcfl)
                {
                    maxcfl = cfl;
                }
            } 
        }
    }            

    __syncthreads();

    ifaces_x = mx + 1;
    ifaces_y = my + 1;
    num_ifaces = ifaces_x*ifaces_y;

    /* Limit waves in the X direction */
    if (order[0] == 2)
    {
        for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
        { 
            ix = thread_index % ifaces_x;
            iy = thread_index/ifaces_y;

            I = (ix + mbc)*xs + (iy + mbc)*ys;

            if (ix < mx + 1 && iy < my + 1)   /* Is this needed? */
            {
                for(mw = 0; mw < mwaves; mw++)
                {
                    I_speeds = I + mw*zs;
                    s[mw] = speeds[I_speeds];

                    for(mq = 0; mq < meqn; mq++)
                    {
                        I_waves = I + (mw*meqn + mq)*zs;
                        wave[mq] = waves[I_waves];
                    }                        

                    if (mthlim[mw] > 0)
                    {
                        wnorm2 = dotl = dotr = 0;
                        for(mq = 0; mq < meqn; mq++)
                        {
                            I_waves = I + (mw*meqn + mq)*zs;
                            wnorm2 += pow(wave[mq],2);
                            dotl += wave[mq]*waves[I_waves-1];
                            dotr += wave[mq]*waves[I_waves+1];
                        }
                        if (wnorm2 != 0)
                        {
                            r = (s[mw] > 0) ? dotl/wnorm2 : dotr/wnorm2;
                            wlimitr = cudaclaw_limiter(mthlim[mw],r);  
                        }
                        for (mq = 0; mq < meqn; mq++)
                        {
                            wave[mq] *= wlimitr;
                        }
                    }

 
                    for(mq = 0; mq < meqn; mq++)
                    {
                        I_q = I + mq*zs;
                        cqxx = fabs(s[mw])*(1.0 - fabs(s[mw])*dtdx)*wave[mq];
                        fm[I_q] += 0.5*cqxx;   
                        fp[I_q] += 0.5*cqxx;                               
                        if (order[1] > 0)
                        {
                            /* Propagate second order corrections 
                               in transverse dir. */
                            amdq_trans[I_q] += cqxx;   
                            apdq_trans[I_q] -= cqxx;      
                        }
                    }
                }
            }
        }
    }

    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    {
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (iy + mbc-1)*ys + (ix + mbc-1)*xs;

        if (ix < mx + 2*mbc-1 && iy < my + 2*mbc-1)
        {
            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                qr[mq] = qold[I_q];        /* Right */
                qd[mq] = qold[I_q - ys];   /* Down  */  
            }

            for(m = 0; m < maux; m++)
            {
                /* How is this getting set? */
                I_aux = I + m*zs;
                auxr[m] = aux[I_aux];
                auxd[m] = aux[I_aux - ys];
            }                        


            /* ---------------------- Normal solver : Y faces ------------------------- */
            rpn2(1, meqn, mwaves, maux, qd, qr, auxd, auxr, wave, s, bmdq, bpdq);

            /* Set value at bottom interface of cell I */
            for (mq = 0; mq < meqn; mq++) 
            {
                I_q = I + mq*zs;
                gm[I_q] = bmdq[mq];
                gp[I_q] = -bpdq[mq]; 
                if (order[1] > 0)
                {
                    bpdq_trans[I_q] = bpdq[mq];
                    bmdq_trans[I_q] = bmdq[mq];                                        
                }
            }

            if (order[0] == 2)
            {
                /* Waves are only needed for second order corrections */
                for (m = 0; m < meqn*mwaves; m++)
                {
                    I_waves = I + (meqn*mwaves+m)*zs;
                    waves[I_waves] = wave[m];
                }                
            }

            for (mw = 0; mw < mwaves; mw++)
            {
                I_speeds = I + (mwaves + mw)*zs;
                speeds[I_speeds] = s[mw];
                cfl = fabs(s[mw])*dtdy;
                if (cfl > maxcfl)
                {
                    maxcfl = cfl;
                }
            } 
        }
    }

    maxcflblocks[blockIdx.z] = BlockReduce(temp_storage).Reduce(maxcfl,hipcub::Max());

    __syncthreads();

    /* ---------------------------------- Limit waves --------------------------------------*/  
    


    ifaces_x = mx + 1;
    ifaces_y = my + 1;
    num_ifaces = ifaces_x*ifaces_y;

    if (order[0] == 2)
    {
        for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
        { 
            ix = thread_index % ifaces_x;
            iy = thread_index/ifaces_y;

            I = (ix + mbc)*xs + (iy + mbc)*ys;

            if (ix < mx + 1 && iy < my + 1)   /* Is this needed? */
            {
                /* Limit waves */
                for(mw = 0; mw < mwaves; mw++)
                {
                    /* X-faces */

                    I_speeds = I + mw*zs;
                    s[mw] = speeds[I_speeds];

                    for(mq = 0; mq < meqn; mq++)
                    {
                        I_waves = I + (mw*meqn + mq)*zs;
                        wave[mq] = waves[I_waves];
                    }                        

                    if (mthlim[mw] > 0)
                    {
                        wnorm2 = dotl = dotr = 0;
                        for(mq = 0; mq < meqn; mq++)
                        {
                            I_waves = I + (mw*meqn + mq)*zs;
                            wnorm2 += pow(wave[mq],2);
                            dotl += wave[mq]*waves[I_waves-1];
                            dotr += wave[mq]*waves[I_waves+1];
                        }
                        if (wnorm2 != 0)
                        {
                            r = (s[mw] > 0) ? dotl/wnorm2 : dotr/wnorm2;
                            wlimitr = cudaclaw_limiter(mthlim[mw],r);  
                        }
                        for (mq = 0; mq < meqn; mq++)
                        {
                            wave[mq] *= wlimitr;
                        }
                    }

 
                    for(mq = 0; mq < meqn; mq++)
                    {
                        I_q = I + mq*zs;
                        cqxx = fabs(s[mw])*(1.0 - fabs(s[mw])*dtdx)*wave[mq];
                        fm[I_q] += 0.5*cqxx;   
                        fp[I_q] += 0.5*cqxx;                               
                        if (order[1] > 0)
                        {
                            /* Propagate second order corrections 
                               in transverse dir. */
                            amdq_trans[I_q] += cqxx;   
                            apdq_trans[I_q] -= cqxx;      
                        }
                    }

                    /* Y-faces */

                    I_speeds = I + (mwaves + mw)*zs;
                    s[mw] = speeds[I_speeds];

                    for(mq = 0; mq < meqn; mq++)
                    {
                        I_waves = I + ((mwaves+mw)*meqn + mq)*zs;
                        wave[mq] = waves[I_waves];
                    }                        

                    if (mthlim[mw] > 0)
                    {
                        wnorm2 = dotl = dotr = 0;
                        for(mq = 0; mq < meqn; mq++)
                        {
                            I_waves = I + ((mwaves+mw)*meqn + mq)*zs;
                            wnorm2 += pow(wave[mq],2);
                            dotl += wave[mq]*waves[I_waves-ys];
                            dotr += wave[mq]*waves[I_waves+ys];
                        }
                        if (wnorm2 != 0)
                        {
                            r = (s[mw] > 0) ? dotl/wnorm2 : dotr/wnorm2;
                            wlimitr = cudaclaw_limiter(mthlim[mw],r);  
                        }
                        for (mq = 0; mq < meqn; mq++)
                        {
                            wave[mq] *= wlimitr;
                        }
                    }

                    for(mq = 0; mq < meqn; mq++)
                    {
                        I_q = I + mq*zs;
                        cqyy = fabs(s[mw])*(1.0 - fabs(s[mw])*dtdy)*wave[mq];
                        gm[I_q] += 0.5*cqyy;   
                        gp[I_q] += 0.5*cqyy;   
                        if (order[1] > 0)
                        {
                            /* Propagate second order corrections 
                               in transverse dir. */
                            bmdq_trans[I_q] += cqyy;     
                            bpdq_trans[I_q] -= cqyy;      
                        }
                    }                
                }  /* End of mwaves loop */
            } /* End of thread conditional */
        } /* End of thread loop */
    } /* End of check on order[0] == 2 */


    __syncthreads();


    if (order[1] == 0)
    {
        /* No transverse propagation; Update the solution and exit */
        for(thread_index = threadIdx.x; thread_index < mx*my; thread_index += blockDim.x)
        {
            ix = thread_index % mx;
            iy = thread_index/my;

            I = (ix + mbc)*xs + (iy + mbc)*ys;

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                qold[I_q] = qold[I_q] - dtdx * (fm[I_q + xs] - fp[I_q]) 
                                      - dtdy * (gm[I_q + ys] - gp[I_q]);
            }        
        }
        return;
    }



    /* ------------------------ Transverse Propagation : X-faces ---------------------- */
    
    /*     transverse-x

            |     |     | 
            |     |     | 
        ----|-----|-----|-----
            |     X     | 
            |  v--X     |
        ----|--O--|-----|-----
            |     |     |
            |     |     |

    */              

    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    { 
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (ix + mbc)*xs + (iy + mbc)*ys;

        /* Lower left face */
        if (0 < ix && ix < mx + 1 && iy < my)   /* Is this needed? */
        {
            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                amdq[mq] = amdq_trans[I_q];
                apdq[mq] = apdq_trans[I_q];
            }            
            
            for(imp = 0; imp < 2; imp++)
            {
                for(m = 0; m < maux; m++)
                {
                    I_aux = I + m*zs;
                    k = imp*maux + m;
                    aux1[k] = aux[I_aux - ys + (imp - 1)];
                    aux2[k] = aux[I_aux      + (imp - 1)];
                    aux3[k] = aux[I_aux + ys + (imp - 1)];
                }
            }

            rpt2(0,meqn,mwaves,maux,ql,qr,aux1,aux2,aux3,0,0,amdq,bmasdq,bpasdq);

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;  
                gupdate = 0.5*dtdx*bmasdq[mq];
                gm[I_q - 1] -= gupdate;       
                gp[I_q - 1] -= gupdate;
            }
        } /* thread conditional */
    } /* Thread loop */

    __syncthreads();

    /*   transverse-x  
            |     |     | 
            |     |     | 
        ----|--O--|-----|----
            |  ^__X     | 
            |     X     |
        ----|-----|-----|----
            |     |     |
            |     |     |

    */              

    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    { 
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (ix + mbc)*xs + (iy + mbc)*ys;

        if (0 < ix && ix < mx + 1 && iy < my)   /* Is this needed? */
        {
            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                amdq[mq] = amdq_trans[I_q];
                apdq[mq] = apdq_trans[I_q];
            }            
            
            for(imp = 0; imp < 2; imp++)
            {
                for(m = 0; m < maux; m++)
                {
                    I_aux = I + m*zs;
                    k = imp*maux + m;
                    aux1[k] = aux[I_aux - ys + (imp - 1)];
                    aux2[k] = aux[I_aux      + (imp - 1)];
                    aux3[k] = aux[I_aux + ys + (imp - 1)];
                }
            }

            rpt2(0,meqn,mwaves,maux,ql,qr,aux1,aux2,aux3,0,1,amdq,bmasdq,bpasdq);

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;  
                gupdate = 0.5*dtdx*bpasdq[mq];
                gm[I_q - 1 + ys] -= gupdate;
                gp[I_q - 1 + ys] -= gupdate;
            }
        } /* thread conditional */
    } /* Thread loop */

    __syncthreads();

    /*  transverse-x
            |     |     | 
            |     |     | 
        ----|-----|-----|----
            |     X     | 
            |     X--v  |
        ----|-----|--O--|----
            |     |     |
            |     |     |

    */              

    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    { 
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (ix + mbc)*xs + (iy + mbc)*ys;

        if (ix < mx && iy < my)   
        {

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                amdq[mq] = amdq_trans[I_q];
                apdq[mq] = apdq_trans[I_q];
            }            
            
            for(imp = 0; imp < 2; imp++)
            {
                for(m = 0; m < maux; m++)
                {
                    I_aux = I + m*zs;
                    k = imp*maux + m;
                    aux1[k] = aux[I_aux - ys + (imp - 1)];
                    aux2[k] = aux[I_aux      + (imp - 1)];
                    aux3[k] = aux[I_aux + ys + (imp - 1)];
                }
            }

            rpt2(0,meqn,mwaves,maux,ql,qr,aux1,aux2,aux3,1,0,apdq,bmasdq,bpasdq);

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;  
                gupdate = 0.5*dtdx*bmasdq[mq];
                gm[I_q] -= gupdate;       
                gp[I_q] -= gupdate;
            }
        } /* thread conditional */
    } /* Thread loop */

    __syncthreads();

    /*  transverse-x 
            |     |     | 
            |     |     | 
        ----|-----|--O--|----
            |     X__^  | 
            |     X     |
        ----|-----|-----|----
            |     |     |
            |     |     |

    */              

    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    { 
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (ix + mbc)*xs + (iy + mbc)*ys;

        if (ix < mx && iy < my)   
        {

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                amdq[mq] = amdq_trans[I_q];
                apdq[mq] = apdq_trans[I_q];
            }            
            
            for(imp = 0; imp < 2; imp++)
            {
                for(m = 0; m < maux; m++)
                {
                    I_aux = I + m*zs;
                    k = imp*maux + m;
                    aux1[k] = aux[I_aux - ys + (imp - 1)];
                    aux2[k] = aux[I_aux      + (imp - 1)];
                    aux3[k] = aux[I_aux + ys + (imp - 1)];
                }
            }

            rpt2(0,meqn,mwaves,maux,ql,qr,aux1,aux2,aux3,1,1,apdq,bmasdq,bpasdq);

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;  
                gupdate = 0.5*dtdx*bpasdq[mq];
                gm[I_q + ys] -= gupdate;
                gp[I_q + ys] -= gupdate;
            }
        } /* thread conditional */
    } /* Thread loop */

    __syncthreads();

    /* ----------------------------- Transverse : Y-faces ----------------------------- */


    /*  transverse-y

             |     |     
             |     |     
        -----|-----|-----
             |     |     
             |     |     
             |     |     
        -----|-XXX-|-----
             |  v  |     
             0--   |     
             |     |     
        -----|-----|-----
             |     |     
             |     |     
    */              

    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    { 
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (ix + mbc)*xs + (iy + mbc)*ys;

        if (ix < mx && 0 < iy && iy < my+1)   /* Is this needed? */
        {

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                bmdq[mq] = bmdq_trans[I_q];
                bpdq[mq] = bpdq_trans[I_q];
            }            

            for(imp = 0; imp < 2; imp++)
            {
                for(m = 0; m < maux; m++)
                {
                    I_aux = I + m*zs;
                    k = imp*maux + m;
                    aux1[k] = aux[I_aux - 1 + ys*(imp - 1)];
                    aux2[k] = aux[I_aux     + ys*(imp - 1)];
                    aux3[k] = aux[I_aux + 1 + ys*(imp - 1)];
                }
            }

            rpt2(1,meqn,mwaves,maux,qd,qr,aux1,aux2,aux3,0,0,bmdq,bmasdq,bpasdq);

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;  
                gupdate = 0.5*dtdy*bmasdq[mq];
                fm[I_q - ys] -= gupdate;        
                fp[I_q - ys] -= gupdate;
            }
        } /* Thread conditional */
    } /* Thread loop */

    __syncthreads();

    /*  transverse-y

             |     |     
             |     |     
        -----|-----|-----
             |     |     
             |     |     
             |     |     
        -----|-XXX-|-----
             |  v  |     
             |   --O     
             |     |     
        -----|-----|-----
             |     |     
             |     |     
    */              

    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    { 
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (ix + mbc)*xs + (iy + mbc)*ys;

        if (ix < mx && 0 < iy && iy < my+1)   /* Is this needed? */
        {
            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                bmdq[mq] = bmdq_trans[I_q];
                bpdq[mq] = bpdq_trans[I_q];
            }            

            for(imp = 0; imp < 2; imp++)
            {
                for(m = 0; m < maux; m++)
                {
                    I_aux = I + m*zs;
                    k = imp*maux + m;
                    aux1[k] = aux[I_aux - 1 + ys*(imp - 1)];
                    aux2[k] = aux[I_aux     + ys*(imp - 1)];
                    aux3[k] = aux[I_aux + 1 + ys*(imp - 1)];
                }
            }

            rpt2(1,meqn,mwaves,maux,qd,qr,aux1,aux2,aux3,0,1,bmdq,bmasdq,bpasdq);

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;  
                gupdate = 0.5*dtdy*bpasdq[mq];
                fm[I_q - ys + 1] -= gupdate;
                fp[I_q - ys + 1] -= gupdate;
            }
        } /* Thread conditional */
    } /* Thread loop */

    __syncthreads();


    /*  transverse-y

             |     |     
             |     |     
        -----|-----|-----
             |     |     
             O---  |     
             |  ^  |     
        -----|-XXX-|-----
             |     |     
             |     |     
             |     |     
        -----|-----|-----
             |     |     
             |     |     
    */ 

    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    { 
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (ix + mbc)*xs + (iy + mbc)*ys;

        if (ix < mx && iy < my)   /* Is this needed? */
        {
            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                bmdq[mq] = bmdq_trans[I_q];
                bpdq[mq] = bpdq_trans[I_q];
            }            

            for(imp = 0; imp < 2; imp++)
            {
                for(m = 0; m < maux; m++)
                {
                    I_aux = I + m*zs;
                    k = imp*maux + m;
                    aux1[k] = aux[I_aux - 1 + ys*(imp - 1)];
                    aux2[k] = aux[I_aux     + ys*(imp - 1)];
                    aux3[k] = aux[I_aux + 1 + ys*(imp - 1)];
                }
            }

            rpt2(1,meqn,mwaves,maux,qd,qr,aux1,aux2,aux3,1,0,bpdq,bmasdq,bpasdq);

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;  
                gupdate = 0.5*dtdy*bmasdq[mq];
                fm[I_q] -= gupdate;        
                fp[I_q] -= gupdate;
            }   
        } /* Thread conditional */
    } /* Thread loop */

    __syncthreads();

    /*  transverse-y

             |     |     
             |     |     
        -----|-----|-----
             |     |     
             |  ---O     
             |  ^  |     
        -----|-XXX-|-----
             |     |     
             |     |     
             |     |     
        -----|-----|-----
             |     |     
             |     |     
    */              

    for(thread_index = threadIdx.x; thread_index < num_ifaces; thread_index += blockDim.x)
    { 
        ix = thread_index % ifaces_x;
        iy = thread_index/ifaces_y;

        I = (ix + mbc)*xs + (iy + mbc)*ys;

        if (ix < mx && iy < my)   /* Is this needed? */
        {

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;
                bmdq[mq] = bmdq_trans[I_q];
                bpdq[mq] = bpdq_trans[I_q];
            }            

            for(imp = 0; imp < 2; imp++)
            {
                for(m = 0; m < maux; m++)
                {
                    I_aux = I + m*zs;
                    k = imp*maux + m;
                    aux1[k] = aux[I_aux - 1 + ys*(imp - 1)];
                    aux2[k] = aux[I_aux     + ys*(imp - 1)];
                    aux3[k] = aux[I_aux + 1 + ys*(imp - 1)];
                }
            }

            rpt2(1,meqn,mwaves,maux,qd,qr,aux1,aux2,aux3,1,1,bpdq,bmasdq,bpasdq);

            for(mq = 0; mq < meqn; mq++)
            {
                I_q = I + mq*zs;  
                gupdate = 0.5*dtdy*bpasdq[mq];
                fm[I_q + 1] -= gupdate;
                fp[I_q + 1] -= gupdate;
            }   
        } /* Thread conditional */
    } /* Thread loop */

    __syncthreads();

    /* ------------------------------- Final update ----------------------------------- */

    for(thread_index = threadIdx.x; thread_index < mx*my; thread_index += blockDim.x)
    {
        ix = thread_index % mx;
        iy = thread_index/my;

        I = (ix + mbc)*xs + (iy + mbc)*ys;

        for(mq = 0; mq < meqn; mq++)
        {
            I_q = I + mq*zs;
            qold[I_q] = qold[I_q] - dtdx * (fm[I_q + xs] - fp[I_q]) 
                                  - dtdy * (gm[I_q + ys] - gp[I_q]);
        }        
    }
}


/* ---------------------------------------------------------------------------------------
   PUBLIC function  
   ------------------------------------------------------------------------------------ */
__global__
void cudaclaw_flux2_and_update_batch (int mx, int my, int meqn, int mbc, 
                                      int maux, int mwaves, int mwork,
                                      double dt, double t,
                                      int* order, int* mthlim,
                                      cudaclaw_fluxes_t* array_fluxes_struct,
                                      double * maxcflblocks,
                                      cudaclaw_cuda_rpn2_t rpn2,
                                      cudaclaw_cuda_rpt2_t rpt2,
                                      cudaclaw_cuda_b4step2_t b4step2)
    {
        cudaclaw_flux2_and_update(mx,my,meqn,mbc,maux,mwaves,mwork,
                                  array_fluxes_struct[blockIdx.z].xlower,
                                  array_fluxes_struct[blockIdx.z].ylower,
                                  array_fluxes_struct[blockIdx.z].dx,
                                  array_fluxes_struct[blockIdx.z].dy,
                                  array_fluxes_struct[blockIdx.z].qold_dev,
                                  array_fluxes_struct[blockIdx.z].aux_dev,
                                  array_fluxes_struct[blockIdx.z].fm_dev,
                                  array_fluxes_struct[blockIdx.z].fp_dev,
                                  array_fluxes_struct[blockIdx.z].gm_dev,
                                  array_fluxes_struct[blockIdx.z].gp_dev,
                                  array_fluxes_struct[blockIdx.z].amdq_dev,
                                  array_fluxes_struct[blockIdx.z].apdq_dev,
                                  array_fluxes_struct[blockIdx.z].bmdq_dev,
                                  array_fluxes_struct[blockIdx.z].bpdq_dev,
                                  array_fluxes_struct[blockIdx.z].waves_dev,
                                  array_fluxes_struct[blockIdx.z].speeds_dev,
                                  maxcflblocks, rpn2, rpt2, b4step2,
                                  order, mthlim, t,dt);
}



